#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <unistd.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples
#include "common.h"

// define constants
#define MAX_LINE 200000
#define MAX_CHAR 40
#define NB_ASCII 128

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void generateHisto(char* inputFileName, char* outputFileName);

void writeOutputCSV(int result[NB_ASCII], char* outputFileName);

void startTimer(StopWatchInterface* timer);

void stopTimer(StopWatchInterface* timer);

////////////////////////////////////////////////////////////////////////////////
//! Kernel function to execute the computation in threads
//! @param d_data  input data in global memory
//! @param d_result  output result as array in global memory
//! @param nbLine  input size of the data in global memory
//! @param pitch  input pitch size of in the data global memory
////////////////////////////////////////////////////////////////////////////////

__global__ 
void kernelFunction(char* d_data, int* d_result, int nbLine, size_t pitch) {
    
    const unsigned int tidb = threadIdx.x;
    const unsigned int ti = blockIdx.x*blockDim.x + tidb;
    
    if (ti < nbLine) {
		char* line = (char *)((char*)d_data + ti * pitch);
		int index = 0;
		int currentLetter = line[index];

		while (currentLetter > 0) {
	    	atomicAdd(&d_result[currentLetter], 1);
	    	index++;
	    	currentLetter = line[index];
		}
    } 
}



////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
	
	int c;
	char *inputFileName = NULL;
	char *outputFileName = NULL;

	while ((c = getopt (argc, argv, "i:o:")) != -1)
		switch(c) {
			case 'i':
				inputFileName = optarg;
				break;
			case 'o':
				outputFileName = optarg;
				break;
			default:
				break;
		}

	printf("%s Starting...\n\n", argv[0]);

	generateHisto(inputFileName, outputFileName);

	exit(EXIT_SUCCESS);
}

////////////////////////////////////////////////////////////////////////////////
//! Generate the Histogram
////////////////////////////////////////////////////////////////////////////////
void generateHisto(char* inputFileName, char* outputFileName) {

    StopWatchInterface *timer = 0;
    startTimer(timer);

    // Print MemInfo
    size_t memfree, memtotal;
    checkCudaErrors(hipMemGetInfo(&memfree, &memtotal));

    unsigned int threadsPerBlock = 1024;
    unsigned int resultSize = NB_ASCII * sizeof(int);
    unsigned int lineSize = MAX_CHAR * sizeof(char);


    // Allocate device memory
    int* d_result;
    char* d_data;
    size_t pitch;
    checkCudaErrors(hipMallocPitch((void **) &d_data, &pitch, lineSize, MAX_LINE));
    checkCudaErrors(hipMalloc((void **) &d_result, resultSize));

    // Setup execution parameters
    dim3  grid((MAX_LINE + threadsPerBlock - 1) / threadsPerBlock, 1, 1);
    dim3  threads(threadsPerBlock, 1, 1);

    // Load input file data
    FILE *inputFile = NULL;
    inputFile = fopen(inputFileName, "r");
    if (!inputFile) {
        printf("Wrong input file\n");
		exit(EXIT_FAILURE);
    }

    // Allocate host memory
    char str[MAX_CHAR];
    char h_data[MAX_LINE][MAX_CHAR];
    int h_result[NB_ASCII];
    int totalResult[NB_ASCII];
    int nbLine = 0;
    
    while (fgets(str, MAX_CHAR, inputFile)) {
	
		if (nbLine == MAX_LINE) {

            printf("Loaded %i lines \n", nbLine);

	    	// Copy data to device
            checkCudaErrors(hipMemcpy2D(d_data, pitch, h_data, lineSize, lineSize, MAX_LINE, hipMemcpyHostToDevice));
            // Execute the kernel
            kernelFunction<<< grid, threads, 0 >>>(d_data, d_result, nbLine, pitch);
            getLastCudaError("Kernel execution failed");
            // Copy result from device to host
            checkCudaErrors(hipMemcpy(&h_result, d_result, resultSize, hipMemcpyDeviceToHost));

            for (int index = 0; index < NB_ASCII; index++) {
                totalResult[index] += h_result[index];
            }
            
            nbLine = 0;
		}

        strcpy(h_data[nbLine], str);
        nbLine++;
    }
    
    printf("Loaded %i lines \n", nbLine);

    // Copy data to device
    checkCudaErrors(hipMemcpy2D(d_data, pitch, h_data, lineSize, lineSize, MAX_LINE, hipMemcpyHostToDevice));
    // Execute the kernel
    kernelFunction<<< grid, threads, 0 >>>(d_data, d_result, nbLine, pitch);
    getLastCudaError("Kernel execution failed");
    // Copy result from device to host
    checkCudaErrors(hipMemcpy(&h_result, d_result, resultSize, hipMemcpyDeviceToHost));

    for (int index = 0; index < NB_ASCII; index++) {
        totalResult[index] += h_result[index];
    } 
    
    fclose(inputFile);
    
    //write the output
    writeOutputCSV(h_result, outputFileName);

    // cleanup memory
    checkCudaErrors(hipFree(d_data));
    checkCudaErrors(hipFree(d_result));

    stopTimer(timer);
}

////////////////////////////////////////////////////////////////////////////////
//! Write the given output to the CSV file
////////////////////////////////////////////////////////////////////////////////

void writeOutputCSV(int result[NB_ASCII], char* outputFileName) {
	FILE *outputFile;
	char asciiChar;

	outputFile = fopen(outputFileName, "w+");
	
	for (int index = 32; index < 127; index++) {
		asciiChar = index;
		fprintf(outputFile, "%c: %i\n", asciiChar, result[index]);
	}

	fclose(outputFile);
}

////////////////////////////////////////////////////////////////////////////////
//! Start the timer
////////////////////////////////////////////////////////////////////////////////
void startTimer(StopWatchInterface* timer) {
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);
}

////////////////////////////////////////////////////////////////////////////////
//! Stop the timer
////////////////////////////////////////////////////////////////////////////////
void stopTimer(StopWatchInterface* timer) {
  sdkStopTimer(&timer);
  printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
  sdkDeleteTimer(&timer);
}
